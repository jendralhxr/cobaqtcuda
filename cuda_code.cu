#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <stdio.h>

extern "C"
hipError_t cuda_main()
{
    printf("stau\n");
    // generate bunch random numbers on the host
    thrust::host_vector<int> h_vec(1 << 25);
    thrust::generate(h_vec.begin(), h_vec.end(), rand);

    printf("dua %ld %ld %ld\n", h_vec.size(), h_vec[20], h_vec[2000]);
    // transfer data to the device
    thrust::device_vector<int> d_vec = h_vec;

    printf("tiga\n");
    // sort data on the device (805 Mkeys/sec on GeForce GTX 480)
    thrust::sort(d_vec.begin(), d_vec.end());

    printf("empat\n");
    // transfer data back to host
    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

    printf("lima %ld %ld %ld\n", h_vec.size(), h_vec[20], h_vec[2000]);
    return hipGetLastError();
}
